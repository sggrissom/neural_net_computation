
#include <iostream>
#include <time.h>
#include <stdlib.h>
// NeuralNet.cpp : Defines the entry point for the console application.

#include "kernel.cu"
#include "support.h"

using std::cout;
using std::endl;

int main(int argc, char* argv[])
{
	enum Mode {CPU_NORMAL = 1, GPU_NAIVE, GPU_IMPROVED};
	Mode mode;
	enum DataSet {XOR = 1};
	DataSet dataSet;

	if(argc == 1) {
		mode = CPU_NORMAL;
		dataSet = XOR;
	} else if(argc == 2) {
		mode = (Mode) atoi(argv[1]);
		dataSet = XOR;
	} else if(argc == 3) {
		mode = (Mode) atoi(argv[1]);
		dataSet = (DataSet) atoi(argv[2]);
	}else {
		printf("\n    Invalid input parameters."
				"\n\n");
		exit(0);
	}

	Timer timer;
	hipError_t cuda_ret;
	printf("\nSetting up the problem..."); fflush(stdout);
	startTime(&timer);

	int inputSize = 3;
	int outputSize = 1;
	int dataPoints = 8;
	int testPoints = 8;

	float *data;
	if (dataSet==XOR) {
		float xorData[]={
			0,	0,	0,	0,
			0,	0,	1,	1,
			0,	1,	0,	1,
			0,	1,	1,	0,
			1,	0,	0,	1,
			1,	0,	1,	0,
			1,	1,	0,	0,
			1,	1,	1,	1 };

		data = xorData;
	}

	int *lsize = new int[4];
	lsize[0] = inputSize;
	lsize[1] = inputSize-1;
	lsize[2] = inputSize-2;
	lsize[3] = outputSize;

	int i;

	float beta = 0.3, alpha = 0.1;
	long num_iter = 2000000;

	float *out;
	float *delta;
	float *weight;
	int numl=4;
	float *prevDwt;

	int numn = 0;
	int rowptr_od[numl+1];
	for(int i=0; i<numl; i++) {
		rowptr_od[i] = numn;
		numn += lsize[i];
	}
	rowptr_od[numl] = numn;

	// Allocate memory for out, delta
	out = new float[numn];
	delta = new float[numn];

	// Allocate memory for weights, prevDwt
	int numw = 0;
	int rowptr_w[numl+1];
	for(int i=0; i<numl-1; i++) {
		rowptr_w[i+1] = numw;
		numw += lsize[i]*lsize[i+1];
	}
	weight = new float[numw];
	prevDwt = new float[numw];

	// Seed and assign random weights; set prevDwt to 0 for first iter
	srand((unsigned)(time(NULL)));
	for(i=1;i<numw;i++) {
		weight[i] = (float)(rand())/(RAND_MAX/2) - 1;//32767
		prevDwt[i] = (float)0.0;
	}

	stopTime(&timer); printf("%f s\n", elapsedTime(timer));

	// Allocate device variables ------------------------------------------

	float *data_d;
	float *out_d;
	float *delta_d;
	int *rowptr_od_d;
	float *weight_d;
	float *prevDwt_d;
	int *rowptr_w_d;
	int *lsize_d;

	if(mode != CPU_NORMAL) {
		printf("Allocating device variables...\n"); fflush(stdout);
		startTime(&timer);

		cuda_ret = hipMalloc((void**)&data_d,
				(inputSize+outputSize)*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&out_d, numn*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&delta_d, numn*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&rowptr_od,
				(numl+1)*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&weight_d, numw*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&prevDwt_d, numw*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&rowptr_w_d,
				(numl+1)*sizeof(float));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");
		cuda_ret = hipMalloc((void**)&lsize_d, numl*sizeof(int));
		if(cuda_ret != hipSuccess)
			FATAL("Unable to allocate device memory");

		hipDeviceSynchronize();
		stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	}

	// Copy host varibles to device ---------------------------------------

	if(mode != CPU_NORMAL) {
		printf("Copying data from host to device...\n");fflush(stdout);
		startTime(&timer);

		cuda_ret = hipMemcpy(data_d, data,
				(inputSize+outputSize)*sizeof(float),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");
		cuda_ret = hipMemcpy(out_d, out, numn*sizeof(float),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");
		cuda_ret = hipMemcpy(delta_d, delta, numn*sizeof(float),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");
		cuda_ret = hipMemcpy(weight_d, weight, numw*sizeof(float),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");
		cuda_ret = hipMemcpy(prevDwt_d, prevDwt, numw*sizeof(float),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");
		cuda_ret = hipMemcpy(rowptr_w_d, rowptr_w, numw*sizeof(float),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");
		cuda_ret = hipMemcpy(lsize_d, lsize, numl*sizeof(int),
				hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess)
			FATAL("Unable to set device memory");

		hipDeviceSynchronize();
		stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	}

	// Launch kernel ------------------------------------------------------

	printf("Launching kernel ");

	if(mode == CPU_NORMAL) {
		printf("(CPU version)...");fflush(stdout);
		startTime(&timer);

		printf("training the network...");
		for (i=0; i<num_iter ; i++)
		{
			cpu_bpgt(&data[(i%dataPoints)*(inputSize+outputSize)],
					&data[(i%dataPoints)*(inputSize+outputSize) + inputSize],
					out,delta,rowptr_od,weight,numl,lsize,beta,
					alpha,prevDwt,rowptr_w);
		}

		stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	} else if(mode == GPU_NAIVE) {
		printf("(GPU naive version)...");fflush(stdout);
		startTime(&timer);

		printf("training the network...");
		for (i=0; i<num_iter ; i++)
		{
			gpu_naive_bpgt(&data_d[(i%dataPoints)*(inputSize+outputSize)],
					&data_d[(i%dataPoints)*(inputSize+outputSize) + inputSize],
					out_d,delta_d,rowptr_od_d,weight_d,numl,lsize_d,beta,
					alpha,prevDwt_d,rowptr_w_d);
		}

		cuda_ret = hipDeviceSynchronize();
		if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");
		stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	} else if(mode == GPU_IMPROVED) {
		printf("(GPU improved version)...");fflush(stdout);
		startTime(&timer);

		printf("training the network...");
		for (i=0; i<num_iter ; i++)
		{
			gpu_improved_bpgt(&data[(i%dataPoints)*(inputSize+outputSize)],
					&data[(i%dataPoints)*(inputSize+outputSize) + inputSize],
					out_d,delta_d,rowptr_od_d,weight_d,numl,lsize_d,beta,
					alpha,prevDwt_d,rowptr_w_d);
		}

		cuda_ret = hipDeviceSynchronize();
		if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");
		stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	} else {
		printf("Invalid mode!\n");
		exit(0);
	}

	// Copy device variables from host ----------------------------------------

	if(mode != CPU_NORMAL) {

		printf("Copying data from device to host..."); fflush(stdout);
		startTime(&timer);

		cuda_ret = hipMemcpy(out, out_d, numn * sizeof(float),
				hipMemcpyDeviceToHost);
		if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

		hipDeviceSynchronize();
		stopTime(&timer); printf("%f s\n", elapsedTime(timer));

	}

	// Verify correctness -----------------------------------------------------



	if ( i == num_iter )
		cout << endl << i << " iterations completed..." << endl;

	cout<< "Now using the trained network to make predctions on test data...." << endl << endl;
	for ( i = 0 ; i < testPoints ; ++i )
	{
		ffwd(&data[i*(inputSize+outputSize)],
				out,weight,numl,lsize);
		for (int j=0; j < inputSize; ++j)
		{
			cout << data[i*(inputSize+outputSize)+j] << " ";
		}
		cout << "Ans:" << data[i*(inputSize+outputSize) + inputSize] <<
		"  Guess:" << out[rowptr_od[numl - 1]] << endl;
	}

	int prediction, actual;
	float guess;
	int correct = 0, incorrect = 0;

	for (int i = 0; i < dataPoints; ++i)
	{
		ffwd(&data[i*(inputSize+outputSize)],
				out,weight,numl,lsize);
		actual = (int) (data[i*(inputSize+outputSize) + inputSize]);
		guess = out[rowptr_od[numl-1]];
		prediction = (int)(guess + 0.5);
		if (prediction == actual)
		{
			correct++;
		} else {
			incorrect++;
		}
	}

	float accuracy = ((float)correct)/((float)(correct+incorrect));

	cout << "Correct = " << correct << endl;
	cout << "Incorrect = " << incorrect << endl;
	cout << "Accuracy = " << accuracy*100 << "%" << endl;


	// free out
	delete[] out;

	// free delta
	delete[] delta;

	// free weight
	delete[] weight;

	// free prevDwt
	delete[] prevDwt;

	// free layer info
	delete[] lsize;

	// Free memory ------------------------------------------------------------

	if(mode != CPU_NORMAL) {
		hipFree(data_d);
		hipFree(out_d);
		hipFree(delta_d);
		hipFree(rowptr_od_d);
		hipFree(weight_d);
		hipFree(prevDwt_d);
		hipFree(rowptr_w_d);
		hipFree(lsize_d);
	}

	return 0;
}
