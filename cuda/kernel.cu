
#include <hip/hip_runtime.h>
#include <math.h>

/*****************************************************************************
  GPU main computation kernels
 *****************************************************************************/


__global__ void gpu_datatest_kernel(double *data,
		double *out,
		double *delta,
		int *rowptr_od,
		double *weight,
		int numl,
		int *lsize,
		double beta,
		double alpha,
		double *prevDwt,
		int *rowptr_w,
		int num_iter,
    int inSize,
    int dataSize) {

  double *in;
  for (int iter=0; iter<num_iter; iter++)
  {
    in = data + (iter%dataSize)*inSize; 

    for (int i = 0; i < inSize; i++) {
        printf("val: %f", in[i]);
    }
    printf("\n");
  }
}


__global__ void gpu_naive_kernel(double *data,
		double *out,
		double *delta,
		int *rowptr_od,
		double *weight,
		int numl,
		int *lsize,
		double beta,
		double alpha,
		double *prevDwt,
		int *rowptr_w,
		int num_iter,
    int inSize,
    int dataSize) {

  double *in = data;
  int i, k, iter;

	for (iter=0; iter<num_iter; iter++)
	{
    in = data + (iter%dataSize)*inSize; 

		int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float sum;
		// update output values for each neuron

		// assign content to input layer
		if (idx < lsize[0])
		{
			// output_from_neuron(i,j) Jth neuron in Ith Layer
			out[idx]=in[idx];
		}

      __syncthreads();

		// assign output(activation) value
		// to each neuron usng sigmoid func
		for (i=1;i<numl;i++)
		{
			if (idx < lsize[i])
			{
        sum=0.0;
				for (k=0;k<lsize[i-1];k++)
				{
					sum += out[rowptr_od[i-1]+k]* weight[rowptr_w[i]+(idx*(lsize[i-1]+1))+k];
				}
				sum += weight[rowptr_w[i]+(lsize[i-1]+1)*idx + lsize[i-1]];
				out[rowptr_od[i]+idx]=(double)(1/(1+exp(-sum)));
			}
		}

		__syncthreads();

		// find delta for output layer
		if (idx == 0)
		{
			for (i=0;i<lsize[numl-1];i++)
			{
				delta[rowptr_od[numl-1]+i]=out[rowptr_od[numl-1]+i]*
          (1-out[rowptr_od[numl-1]+i])*(in[lsize[0]]-out[rowptr_od[numl-1]+i]);
			}
		}

      __syncthreads();

		//	find delta for hidden layers
		for (i=numl-2;i>0;i--)
		{
			if (idx<lsize[i])
			{
        sum=0.0;
				for (k=0;k<lsize[i+1];k++)
				{
					sum += delta[rowptr_od[i+1]+k]*weight[rowptr_w[i+1]+k*(lsize[i]+1)+idx];
				}
				delta[rowptr_od[i]+idx]=out[rowptr_od[i]+idx]*(1-out[rowptr_od[i]+idx])*sum;
        __syncthreads();
			}
		}

		__syncthreads();

		//	apply momentum ( does nothing if alpha=0 )
		for (i=1;i<numl;i++)
		{
			if (idx<lsize[i])
			{
				for (k=0;k<lsize[i-1];k++)
				{
					weight[rowptr_w[i]+idx*(lsize[i-1]+1)+k]+=alpha*prevDwt[rowptr_w[i]+idx*(lsize[i-1]+1)+k];
				}
				weight[rowptr_w[i]+idx*(lsize[i-1]+1)+lsize[i-1]]+=alpha*prevDwt[rowptr_w[i]+idx*(lsize[i-1]+1)+lsize[i-1]];
			}
			__syncthreads();
		}
		__syncthreads();

		//	adjust weights usng steepest descent
		for (i=1;i<numl;i++)
		{
			if (idx<lsize[i])
			{
				for (k=0;k<lsize[i-1];k++)
				{
					prevDwt[rowptr_w[i]+idx*(lsize[i-1]+1)+k]=beta*delta[rowptr_od[i]+idx]*out[rowptr_od[i-1]+k];
					weight[rowptr_w[i]+idx*(lsize[i-1]+1)+k]+=prevDwt[rowptr_w[i]+idx*(lsize[i-1]+1)+k];
				}
				prevDwt[rowptr_w[i]+idx*(lsize[i-1]+1)+lsize[i-1]]=beta*delta[rowptr_od[i]+idx];
				weight[rowptr_w[i]+idx*(lsize[i-1]+1)+lsize[i-1]]+=prevDwt[rowptr_w[i]+idx*(lsize[i-1]+1)+lsize[i-1]];
			}
		}
	}
}

/*****************************************************************************
  Main computation functions
 *****************************************************************************/

void gpu_datatest(double *in,
		double *out,
		double *delta,
		int *rowptr_od,
		double *weight,
		int numl,
		int *lsize,
		double beta,
		double alpha,
		double *prevDwt,
		int *rowptr_w,
		int num_iter,
    int inSize,
    int dataSize) {

        printf("c'mon");
	const unsigned int numThreadsPerBlock = 1;
	const unsigned int numBlocks = 1;
	gpu_datatest_kernel <<< numBlocks , numThreadsPerBlock >>>
		(in,out,delta,rowptr_od,weight,numl,lsize,beta,alpha,
		prevDwt,rowptr_w,num_iter, inSize, dataSize);
}

void gpu_naive_bpgt(double *in,
		double *out,
		double *delta,
		int *rowptr_od,
		double *weight,
		int numl,
		int *lsize,
		double beta,
		double alpha,
		double *prevDwt,
		int *rowptr_w,
		int num_iter,
    int inSize,
    int dataSize) {

	const unsigned int numThreadsPerBlock = 512;
	const unsigned int numBlocks = (128 - 1)/numThreadsPerBlock + 1;
	gpu_naive_kernel <<< numBlocks , numThreadsPerBlock >>>
		(in,out,delta,rowptr_od,weight,numl,lsize,beta,alpha,
		prevDwt,rowptr_w,num_iter,inSize, dataSize);
}

void cpu_bpgt(double *in,double *tgt,
		double *out,
		double *delta,
		int *rowptr_od,
		double *weight,
		int numl,
		int *lsize,
		double beta,
		double alpha,
		double *prevDwt,
		int *rowptr_w)
{
	double sum;
	int i,j,k;

	for (i=0;i<lsize[0];i++)
	{
		out[rowptr_od[0]+i]=in[i];
	}

	for (i=1;i<numl;i++)
	{
		for (j=0;j<lsize[i];j++)
		{
			sum=0.0;
			for (k=0;k<lsize[i-1];k++)
			{
				sum+= out[rowptr_od[i-1]+k]*weight[rowptr_w[i] + (lsize[i-1]+1)*j+k];
			}
			sum+=weight[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]];
			out[rowptr_od[i]+j]=(double)(1/(1+exp(-sum)));
		}
	}

	for (i=0;i<lsize[(numl)-1];i++)
	{
		delta[rowptr_od[(numl)-1]+i]=out[rowptr_od[(numl)-1]+i]*
			(1-out[rowptr_od[(numl)-1]+i])*(tgt[i]-out[rowptr_od[(numl)-1]+i]);
	}

	for (i=numl-2;i>0;i--)
	{
		for (j=0;j<lsize[i];j++)
		{
			sum=0.0;
			for (k=0;k<lsize[i+1];k++)
			{
				sum+=delta[rowptr_od[i+1]+k]*weight[rowptr_w[i+1]+(lsize[i]+1)*k+j];
			}
			delta[rowptr_od[i]+j]=out[rowptr_od[i]+j]*(1-out[rowptr_od[i]+j])*sum;
		}
	}

	for (i=1;i<numl;i++)
	{
		for (j=0;j<lsize[i];j++)
		{
			for (k=0;k<lsize[i-1];k++)
			{
				weight[rowptr_w[i] + (lsize[i-1]+1)*j+k]+=(alpha)*prevDwt[rowptr_w[i] + (lsize[i-1]+1)*j+k];
			}
			weight[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]]+=(alpha)*prevDwt[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]];
		}
	}

	for (i=1;i<numl;i++)
	{
		for (j=0;j<lsize[i];j++)
		{
			for (k=0;k<lsize[i-1];k++)
			{
				prevDwt[rowptr_w[i] + (lsize[i-1]+1)*j+k]=(beta)*delta[rowptr_od[i]+j]*out[rowptr_od[i-1]+k];
				weight[rowptr_w[i] + (lsize[i-1]+1)*j+k]+=prevDwt[rowptr_w[i] + (lsize[i-1]+1)*j+k];
			}
			prevDwt[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]]=(beta)*delta[rowptr_od[i]+j];
			weight[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]]+=prevDwt[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]];
		}
	}
}

void ffwd(double *in,
		double *out,
		double *weight,
		int numl,
		int *lsize,
		int *rowptr_od,
		int *rowptr_w)
{
	double sum;
	int i,j,k;

	for (i=0;i<lsize[0];i++)
	{
		out[rowptr_od[0]+i]=in[i];
	}

	for (i=1;i<numl;i++)
	{
		for (j=0;j<lsize[i];j++)
		{
			sum=0.0;
			for (k=0;k<lsize[i-1];k++)
			{
				sum+= out[rowptr_od[i-1]+k]*weight[rowptr_w[i]
						+ (lsize[i-1]+1)*j+k];
			}
			sum+=weight[rowptr_w[i] + (lsize[i-1]+1)*j+lsize[i-1]];
			out[rowptr_od[i]+j]=(double)(1/(1+exp(-sum)));
		}
	}
}

